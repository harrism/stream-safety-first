#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN

#include <rmm/mr/device/cuda_async_memory_resource.hpp>

#include <catch2/catch.hpp>

#include <algorithm>
#include <numeric>
#include <vector>

namespace {

__global__ void kernel(int* input, int* output, int n, int iterations = 1000)
{
  auto idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) {
    for (int i = 0; i < iterations; i++) {
      output[idx] = input[idx] * input[idx] + i;
    }
  }
}

class device_buffer {
 public:
  device_buffer(std::size_t size, hipStream_t stream) : _size(size), _stream(stream)
  {
    hipMallocAsync(&_data, _size, _stream);
  }

  ~device_buffer() { hipFreeAsync(_data, _stream); }

  void* data() { return _data; }

 private:
  void* _data{};
  std::size_t _size;
  hipStream_t _stream;
};

}  // namespace

class DataRaceFixture {
 protected:
  DataRaceFixture()
  {
    int device{};
    hipGetDevice(&device);
    // Construct explicit pool
    hipMemPoolProps pool_props{};
    pool_props.allocType     = hipMemAllocationTypePinned;
    pool_props.handleTypes   = hipMemHandleTypePosixFileDescriptor;
    pool_props.location.type = hipMemLocationTypeDevice;
    pool_props.location.id   = device;
    hipMemPoolCreate(&cuda_pool_handle_, &pool_props);
    std::size_t free{};
    std::size_t total{};
    hipMemGetInfo(&free, &total);
    hipMemPoolSetAttribute(cuda_pool_handle_, hipMemPoolAttrReleaseThreshold, &total);
    hipDeviceSetMemPool(device, cuda_pool_handle_);

    // initialize input to sequence [0, n)
    h_input.resize(n);
    std::iota(h_input.begin(), h_input.end(), 0);
    // page-lock the host data to enable asynchronous copies
    hipHostRegister(h_input.data(), bytes, hipHostRegisterDefault);

    // initialize output to zeros
    h_output.resize(n);
    // page-lock the host data to enable asynchronous copies
    hipHostRegister(h_output.data(), bytes, hipHostRegisterDefault);

    // generate host reference vector where each element is the square of the
    // input + 999
    h_reference.resize(n);
    std::transform(
      h_input.begin(), h_input.end(), h_reference.begin(), [](int x) { return (x * x) + 999; });

    hipStreamCreate(&stream_a);
    hipStreamCreate(&stream_b);

    hipEventCreateWithFlags(&event_a, hipEventDisableTiming);

    int* big;
    hipMallocAsync(&big, 0.9 * total, stream_a);
    hipFreeAsync(big, stream_a);
  }

  ~DataRaceFixture()
  {
    hipStreamDestroy(stream_a);
    hipStreamDestroy(stream_b);

    hipHostUnregister(h_output.data());
    hipHostUnregister(h_input.data());

    hipEventDestroy(event_a);
  }

  int n{1 << 20};
  int block_sz = 256;
  int num_blocks{(n + block_sz - 1) / block_sz};
  std::size_t bytes{n * sizeof(int)};

  std::vector<int> h_input{};
  std::vector<int> h_output{};
  std::vector<int> h_reference{};

  int* input{};
  int* output{};
  int* foo{};
  int* bar{};

  hipStream_t stream_a{};
  hipStream_t stream_b{};
  hipEvent_t event_a{};

  hipMemPool_t cuda_pool_handle_{};
};

TEST_CASE_METHOD(DataRaceFixture, "Simple data race", "[example_1]")
{
  hipMalloc(&input, bytes);
  hipMalloc(&foo, bytes);
  hipMalloc(&bar, bytes);

  hipMemcpyAsync(input, h_input.data(), bytes, hipMemcpyDefault, stream_a);

  SECTION("Unsafe: Data race between kernel and hipMemcpyAsync")
  {
    kernel<<<num_blocks, block_sz, 0, stream_a>>>(input, foo, n);
    kernel<<<num_blocks, block_sz, 0, stream_a>>>(input, bar, n);
    // data race: possible read of `foo` on `stream_b` before `kernel` writes to it on `stream_a`
    hipMemcpyAsync(h_output.data(), foo, bytes, hipMemcpyDefault, stream_b);

    hipStreamSynchronize(stream_b);
    REQUIRE(h_output != h_reference);
  }

  SECTION("Safe: No data race using a hipStreamWaitEvent")
  {
    kernel<<<num_blocks, block_sz, 0, stream_a>>>(input, foo, n);
    hipEventRecord(event_a, stream_a);
    kernel<<<num_blocks, block_sz, 0, stream_a>>>(input, bar, n);
    // prevent data race by waiting on event
    hipStreamWaitEvent(stream_b, event_a);
    hipMemcpyAsync(h_output.data(), foo, bytes, hipMemcpyDefault, stream_b);

    hipStreamSynchronize(stream_b);
    REQUIRE(h_output == h_reference);
  }

  hipDeviceSynchronize();

  hipFree(bar);
  hipFree(foo);
  hipFree(input);
}

TEST_CASE_METHOD(DataRaceFixture, "Use after free", "[example_2]")
{
  hipMallocAsync(&input, bytes, stream_a);
  hipMallocAsync(&foo, bytes, stream_a);
  hipMallocAsync(&bar, bytes, stream_a);

  hipMemcpyAsync(input, h_input.data(), bytes, hipMemcpyDefault, stream_a);

  SECTION("Unsafe: use-after-free of foo on stream_b")
  {
    kernel<<<num_blocks, block_sz, 0, stream_a>>>(input, foo, n);
    hipEventRecord(event_a, stream_a);
    kernel<<<num_blocks, block_sz, 0, stream_a>>>(input, bar, n, 1);
    // prevent data race by waiting on event
    hipStreamWaitEvent(stream_b, event_a);
    hipMemcpyAsync(h_output.data(), foo, bytes, hipMemcpyDefault, stream_b);

    hipFreeAsync(bar, stream_a);
    // use-after-free of foo on stream_b
    hipFreeAsync(foo, stream_a);
    hipFreeAsync(input, stream_a);

    // This exercises the use-after-free. It is not guaranteed to reproduce on all systems.
    // However, on CUDA 11.5 with a Quadro GV100 (16GB) the memory allocated overlaps foo and
    // the allocation and memset are fast enough to overlap the `hipMemcpyAsync` on `stream_b`
    // above
    int* racer{};
    hipMallocAsync(&racer, 100 * bytes, stream_a);
    hipMemsetAsync(racer, 0xcc, 100 * bytes, stream_a);
    hipFreeAsync(racer, stream_a);

    hipStreamSynchronize(stream_b);
    REQUIRE(h_output != h_reference);
  }

  SECTION("Safe: Free foo on stream_b where last used")
  {
    kernel<<<num_blocks, block_sz, 0, stream_a>>>(input, foo, n);
    hipEventRecord(event_a, stream_a);
    kernel<<<num_blocks, block_sz, 0, stream_a>>>(input, bar, n, 1);
    // prevent data race by waiting on event
    hipStreamWaitEvent(stream_b, event_a);
    hipMemcpyAsync(h_output.data(), foo, bytes, hipMemcpyDefault, stream_b);

    hipFreeAsync(bar, stream_a);
    // No use-after-free of foo on stream_b since we free it on stream_b
    hipFreeAsync(foo, stream_b);
    hipFreeAsync(input, stream_a);

    // Since there is no use-after-free, this code cannot overwrite the contents of `foo` as in
    // the `UseAfterFree` test.
    int* racer{};
    hipMallocAsync(&racer, 100 * bytes, stream_a);
    hipMemsetAsync(racer, 0xcc, 100 * bytes, stream_a);
    hipFreeAsync(racer, stream_a);

    hipStreamSynchronize(stream_b);
    REQUIRE(h_output == h_reference);
  }
}

TEST_CASE_METHOD(DataRaceFixture, "device_buffer use-after-free", "[example_3]")
{
  SECTION("Unsafe: RAII device_buffer use after free")
  {
    device_buffer output(bytes, stream_a);

    {
      device_buffer input(bytes, stream_a);
      hipMemcpyAsync(input.data(), h_input.data(), bytes, hipMemcpyDefault, stream_a);
      hipStreamSynchronize(stream_a);
      kernel<<<num_blocks, block_sz, 0, stream_b>>>(
        static_cast<int*>(input.data()), static_cast<int*>(output.data()), n);
    }

    // input is out of scope and therefore its memory could be reused on stream_a
    // meanwhile kernel may still be reading from it on stream_b...

    {
      // This exercises the use-after-free. It is not guaranteed to reproduce on all systems.
      // However, on CUDA 11.5 with a Quadro GV100 (16GB) the memory allocated overlaps foo and
      // the allocation and memset are fast enough to overlap the `kernel` on `stream_b`
      // above
      device_buffer racer(100 * bytes, stream_a);
      hipMemsetAsync(racer.data(), 0xcc, 100 * bytes, stream_a);
    }

    hipMemcpyAsync(h_output.data(), output.data(), bytes, hipMemcpyDefault, stream_b);
    hipStreamSynchronize(stream_b);
    REQUIRE(h_output == h_reference);  // Technically this could fail
  }

  SECTION("Safe: synchronize streams before and after cross-stream use.")
  {
    device_buffer output(bytes, stream_a);

    {
      device_buffer input(bytes, stream_a);
      hipMemcpyAsync(input.data(), h_input.data(), bytes, hipMemcpyDefault, stream_a);
      hipStreamSynchronize(stream_a);
      kernel<<<num_blocks, block_sz, 0, stream_b>>>(
        static_cast<int*>(input.data()), static_cast<int*>(output.data()), n);
      hipStreamSynchronize(stream_b);
    }

    // input is out of scope, but only after the kernel finished writing to output.

    {
      // Since there is no use-after-free, this code cannot overwrite the contents of `output` as in
      // the `UseAfterFree` test.
      device_buffer racer(100 * bytes, stream_a);
      hipMemsetAsync(racer.data(), 0xcc, 100 * bytes, stream_a);
    }

    hipMemcpyAsync(h_output.data(), output.data(), bytes, hipMemcpyDefault, stream_a);
    hipStreamSynchronize(stream_a);
    REQUIRE(h_output == h_reference);
  }

  SECTION("Safe: RAII device_buffer used on same stream as it is freed")
  {
    device_buffer output(bytes, stream_a);

    {
      device_buffer input(bytes, stream_a);
      hipMemcpyAsync(input.data(), h_input.data(), bytes, hipMemcpyDefault, stream_a);
      kernel<<<num_blocks, block_sz, 0, stream_a>>>(
        static_cast<int*>(input.data()), static_cast<int*>(output.data()), n);
    }

    // input is out of scope, but kernel and memcpy ran on the same stream so no synchronization
    // necessary

    {
      // Since there is no use-after-free, this code cannot overwrite the contents of `output` as in
      // the `UseAfterFree` test.
      device_buffer racer(100 * bytes, stream_a);
      hipMemsetAsync(racer.data(), 0xcc, 100 * bytes, stream_a);
    }

    hipMemcpyAsync(h_output.data(), output.data(), bytes, hipMemcpyDefault, stream_a);
    hipStreamSynchronize(stream_a);
    REQUIRE(h_output == h_reference);
  }
}
